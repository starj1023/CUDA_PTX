#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdint>
#include <time.h>


//Simeck-64/128
__global__ void simeck(uint32_t *keys, uint32_t*ciphertext)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t temp;
    uint32_t constant = 0xFFFFFFFC;
    uint64_t sequence = 0x938BCA3083F;
    uint32_t temp_result[2];
    uint32_t result[2];

    for (int i = 0; i < 44; i++) {
        //Round function
        asm("{\n\t"
            "mov.u32 %0, %1; \n\t"
            "shl.b32 %2, %1, 5; \n\t"
            "shr.b32 %3, %1, 27; \n\t"
            "or.b32 %4, %2, %3; \n\t"
            "shl.b32 %2, %1, 1; \n\t"
            "shr.b32 %3, %1, 31; \n\t"
            "or.b32 %5, %2, %3; \n\t"
            "and.b32 %1, %1, %4; \n\t"
            "xor.b32 %1, %1, %5; \n\t"
            "xor.b32 %1, %1, %6; \n\t"
            "xor.b32 %1, %1, %7; \n\t"
            "mov.u32 %6, %0; \n\t"
            "}"
            : "+r"(temp), "+r"(ciphertext[2 * k + 1]), "+r"(temp_result[0]) "+r"(temp_result[1]), "+r"(result[0]), "+r"(result[1]), "+r"(ciphertext[2 * k]), "+r"(keys[4 * k])
        );

        constant &= 0xFFFFFFFC;
        constant |= sequence & 1;
        sequence >>= 1;

        //Keyschedule
        asm("{\n\t"
            "mov.u32 %0, %1; \n\t"
            "shl.b32 %2, %1, 5; \n\t"
            "shr.b32 %3, %1, 27; \n\t"
            "or.b32 %4, %2, %3; \n\t"
            "shl.b32 %2, %1, 1; \n\t"
            "shr.b32 %3, %1, 31; \n\t"
            "or.b32 %5, %2, %3; \n\t"
            "and.b32 %1, %1, %4; \n\t"
            "xor.b32 %1, %1, %5; \n\t"
            "xor.b32 %1, %1, %6; \n\t"
            "xor.b32 %1, %1, %7; \n\t"
            "mov.u32 %6, %0; \n\t"
            "mov.u32 %0, %1; \n\t"
            "mov.u32 %1, %8; \n\t"
            "mov.u32 %8, %9; \n\t"
            "mov.u32 %9, %0; \n\t"
            "}"
            : "+r"(temp), "+r"(keys[4 * k + 1]), "+r"(temp_result[0]) "+r"(temp_result[1]), "+r"(result[0]), "+r"(result[1]), "+r"(keys[4 * k]), "+r"(constant), "+r"(keys[4 * k + 2]), "+r"(keys[4 * k + 3])
        );
    }
}



//Smion-64/128
__global__ void simon(uint32_t* K, uint32_t* Pt) {
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    uint32_t c = 0xfffffffc;
    uint64_t z = 0xfc2ce51207a635db;
    uint32_t rk[44];
    rk[0] = K[4 * k]; rk[1] = K[4 * k + 1]; rk[2] = K[4 * k + 2]; rk[3] = K[4 * k + 3];

    uint32_t temp_result[2];
    uint32_t result[2];
    uint8_t i;

    //Keyschedule
    for (i = 4; i < 44; i++) {
        asm("{\n\t"
            "xor.b32 %2, %0, %1; \n\t" 
            "shr.b32 %4, %3, 3; \n\t"
            "shl.b32 %5, %3, 29; \n\t"
            "or.b32 %4, %4, %5; \n\t"
            "xor.b32 %2, %2, %4; \n\t"
            "xor.b32 %2, %2, %6; \n\t"
            "shr.b32 %4, %3, 4; \n\t"
            "shl.b32 %5, %3, 28; \n\t"
            "or.b32 %4, %4, %5; \n\t"
            "xor.b32 %2, %2, %4; \n\t"
            "shr.b32 %4, %6, 1; \n\t"
            "shl.b32 %5, %6, 31; \n\t"
            "or.b32 %4, %4, %5; \n\t"
            "xor.b32 %2, %2, %4; \n\t"
            "}"
            : "+r"(c), "+r"(rk[i - 4]), "+r"(rk[i]), "+r"(rk[i - 1]), "+r"(temp_result[0]), "+r"(temp_result[1]), "+r"(rk[i - 3])
        );
        rk[i] ^= (z & 1);
        z >>= 1;
    }

    ////Round function
    for (i = 0; i < 44; i = i + 2) {
        asm("{\n\t"
            "shl.b32 %1, %0, 1; \n\t"
            "shr.b32 %2, %0, 31; \n\t"
            "or.b32 %3, %1, %2; \n\t"
            "shl.b32 %1, %0, 8; \n\t"
            "shr.b32 %2, %0, 24; \n\t"
            "or.b32 %4, %1, %2; \n\t"
            "and.b32 %3, %3, %4; \n\t"
            "xor.b32 %5, %5, %3; \n\t"
            "shl.b32 %1, %0, 2; \n\t"
            "shr.b32 %2, %0, 30; \n\t"
            "or.b32 %1, %1, %2; \n\t"
            "xor.b32 %5, %5, %1; \n\t"
            "xor.b32 %5, %5, %6; \n\t"
            "shl.b32 %1, %5, 1; \n\t"
            "shr.b32 %2, %5, 31; \n\t"
            "or.b32 %3, %1, %2; \n\t"
            "shl.b32 %1, %5, 8; \n\t"
            "shr.b32 %2, %5, 24; \n\t"
            "or.b32 %4, %1, %2; \n\t"
            "and.b32 %3, %3, %4; \n\t"
            "xor.b32 %0, %0, %3; \n\t"
            "shl.b32 %1, %5, 2; \n\t"
            "shr.b32 %2, %5, 30; \n\t"
            "or.b32 %1, %1, %2; \n\t"
            "xor.b32 %0, %0, %1; \n\t"
            "xor.b32 %0, %0, %7; \n\t"
            "}"
            : "+r"(Pt[2 * k + 1]), "+r"(temp_result[0]), "+r"(temp_result[1]), "+r"(result[0]), "+r"(result[1]), "+r"(Pt[2 * k]), "+r"(rk[i]), "+r"(rk[i + 1])
            );
    }
}

//Speck-64128
__global__ void speck(uint32_t *K, uint32_t *Pt) {
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    uint32_t temp_result[2];
    uint32_t result[2];
    uint32_t rk[27];
    uint32_t i;

    //Keyschedule
    uint32_t A = K[4 * k], B = K[4 * k + 1], C = K[4 * k + 2], D = K[4 * k + 3];
    for (i = 0; i<27;) {
        asm("{\n\t"\
            "mov.u32 %4, %0; \n\t"
            "shr.b32 %1, %3, 8; \n\t"
            "shl.b32 %2, %3, 24; \n\t"
            "or.b32 %3, %1, %2; \n\t"
            "add.u32 %3, %3, %0; \n\t"
            "xor.b32 %3, %3, %5; \n\t"
            "shl.b32 %1, %0, 3; \n\t"
            "shr.b32 %2, %0, 29; \n\t"
            "or.b32 %0, %1, %2; \n\t"
            "xor.b32 %0, %0, %3; \n\t" 
            "add.u32 %5, %5, 1; \n\t"
            "mov.u32 %8, %0; \n\t" 
            "shr.b32 %1, %6, 8; \n\t" 
            "shl.b32 %2, %6, 24; \n\t"
            "or.b32 %6, %1, %2; \n\t"
            "add.u32 %6, %6, %0; \n\t"
            "xor.b32 %6, %6, %5; \n\t"
            "shl.b32 %1, %0, 3; \n\t"
            "shr.b32 %2, %0, 29; \n\t"
            "or.b32 %0, %1, %2; \n\t"
            "xor.b32 %0, %0, %6; \n\t"
            "add.u32 %5, %5, 1; \n\t"
            "mov.u32 %9, %0; \n\t"
            "shr.b32 %1, %7, 8; \n\t"
            "shl.b32 %2, %7, 24; \n\t"
            "or.b32 %7, %1, %2; \n\t"
            "add.u32 %7, %7, %0; \n\t"
            "xor.b32 %7, %7, %5; \n\t"
            "shl.b32 %1, %0, 3; \n\t"
            "shr.b32 %2, %0, 29; \n\t"
            "or.b32 %0, %1, %2; \n\t"
            "xor.b32 %0, %0, %7; \n\t"
            "add.u32 %5, %5, 1; \n\t" 
            "}"
            : "+r"(A), "+r"(temp_result[0]), "+r"(temp_result[1]), "+r"(B), "+r"(rk[i]), "+r"(i), "+r"(C), "+r"(D), "+r"(rk[i+1]), "+r"(rk[i + 2])
           );
    }

    ////Round function
    for (i = 0; i < 27; i++) {
        asm("{\n\t"
            "shr.b32 %1, %0, 8; \n\t"
            "shl.b32 %2, %0, 24; \n\t"
            "or.b32 %0, %1, %2; \n\t"
            "add.u32 %0, %0, %3; \n\t"
            "xor.b32 %0, %0, %4; \n\t"
            "shl.b32 %1, %3, 3; \n\t"
            "shr.b32 %2, %3, 29; \n\t"
            "or.b32 %3, %1, %2; \n\t"
            "xor.b32 %3, %3, %0; \n\t"
            "}"
            : "+r"(Pt[2 * k + 1]), "+r"(temp_result[0]), "+r"(temp_result[1]), "+r"(Pt[2 * k]), "+r"(rk[i])
        );
    }
    
}


int main() {
    int blocknum = 35;
    int number = 1024 * blocknum;
    uint32_t text[1024 * 35][2];
    uint32_t key[1024 * 35][4];
    uint32_t* d_text, * d_key;
    time_t start, end;

    for (int i = 0; i < number; i++) {
        text[i][0] = 0xffffffff;
        text[i][1] = 0xffffffff;
        key[i][0] = 0xffffffff;
        key[i][1] = 0xffffffff;
        key[i][2] = 0xffffffff;
        key[i][3] = 0xffffffff;
    }
   
    hipMalloc((void**)&d_text, sizeof(uint32_t) * number * 2);
    hipMalloc((void**)&d_key, sizeof(uint32_t) * number * 4);

    start = clock();
    for (int i = 0; i < 100; i++) {
        hipMemcpy(d_text, text, sizeof(uint32_t) * number * 2, hipMemcpyHostToDevice);
        hipMemcpy(d_key, key, sizeof(uint32_t) * number * 4, hipMemcpyHostToDevice);

        simeck << < blocknum, number / blocknum >> > (d_key, d_text);
        //simon << < blocknum, number / blocknum >> > (d_key, d_text);
        //speck << < blocknum, number / blocknum >> > (d_key, d_text);

        hipMemcpy(text, d_text, sizeof(uint32_t) * number * 2, hipMemcpyDeviceToHost);
    }
    end = clock();

    printf("Elapsed time : %f\n", (double(end - start)/CLOCKS_PER_SEC)/100);
    return 0;
}
